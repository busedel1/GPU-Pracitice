#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void Kernel

(
    double* u1,
    double* v1,
    double a,
    double b,
    double eta,
    double d_u1,
    double d_v1,
    double dt,
    double D,
    int N
)

{
      
    int tidx = threadIdx.x;
    int tidy = threadIdx.y;
    int bidx = blockIdx.x;
    int bidy = blockIdx.y;
    int bdimx = blockDim.x;
    int bdimy = blockDim.y;

    int x = tidx+bidx*bdimx;
    int y = tidy+bidy*bdimy;
    int id = x + y*N;
    
    double u1_c,v1_c;
    
    u1_c=u1[id];
    v1_c=v1[id];

    //lap_Du1R
    double u1_l,u1_r,u1_u,u1_d,lap_Du1;
    if(x==0)   u1_l=u1[id+(N-1)];
    else       u1_l=u1[id-1]; 
    if(x==N-1) u1_r=u1[id-(N-1)];
    else       u1_r=u1[id+1];
    if(y==0)   u1_u=u1[id+N*(N-1)];
    else       u1_u=u1[id-N];
    if(y==N-1) u1_d=u1[id-N*(N-1)];
    else       u1_d=u1[id+N];
    lap_Du1=u1_l+u1_r+u1_u+u1_d-u1_c*4.0;

    //lap_Du1I
    double v1_l,v1_r,v1_u,v1_d,lap_Dv1;
    if(x==0)   v1_l=v1[id+(N-1)];
    else       v1_l=v1[id-1]; 
    if(x==N-1) v1_r=v1[id-(N-1)];
    else       v1_r=v1[id+1];
    if(y==0)   v1_u=v1[id+N*(N-1)];
    else       v1_u=v1[id-N]; 
    if(y==N-1) v1_d=v1[id-N*(N-1)];
    else       v1_d=v1[id+N];
    lap_Dv1=v1_l+v1_r+v1_u+v1_d-v1_c*4.0;


    //reaction
    double react_u1,react_v1;
    react_u1=eta*( a-(b+1.0)*u1_c + u1_c*u1_c*v1_c );
    react_v1=eta*( b*u1_c - u1_c*u1_c*v1_c );

    //main
    u1[id] = u1_c+dt*react_u1+d_u1*D*lap_Du1;
    v1[id] = v1_c+dt*react_v1+d_v1*D*lap_Dv1;
  

}
